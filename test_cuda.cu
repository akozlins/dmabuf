#include "hip/hip_runtime.h"
//

#include "test.h"

inline
void cuda_assert(hipError_t hipError_t, const char* function, const char* file, int line, bool abort = true) {
    if(hipError_t == hipSuccess) return;

    fprintf(stderr, "F [%s] %s:%d, hipError_t = %d (%s)\n", function, file, line, hipError_t, hipGetErrorString(hipError_t));
    if(abort) exit(EXIT_FAILURE);
}

#define CUDA_ASSERT(hipError_t) do { cuda_assert((hipError_t), __FUNCTION__, __FILE__, __LINE__); } while(0)

struct cuda_t {
    int device = 0;

    hipDeviceProp_t properties;

    cuda_t() {
        CUDA_ASSERT(hipSetDevice(device));
        CUDA_ASSERT(hipGetDeviceProperties(&properties, device));
    }
};

__global__
void kernel1(uint32_t* values) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    values[i] = ~values[i];
}

__host__
int main() {
    cuda_t cuda;
    CUDA_ASSERT(hipSetDeviceFlags(hipDeviceMapHost));

    INFO("pageableMemoryAccess = %d\n", cuda.properties.pageableMemoryAccess);
    int hostRegisterSupported = 0;
    CUDA_ASSERT(hipDeviceGetAttribute(&hostRegisterSupported, hipDeviceAttributeHostRegisterSupported, cuda.device));
    INFO("hostRegisterSupported = %d\n", hostRegisterSupported);

    test_t test;
    ssize_t size = test.seek_end(), offset = 0;
    test.mmap(size, offset);

    int nThreadsPerBlock = 1;
    while(2 * nThreadsPerBlock <= cuda.properties.maxThreadsPerBlock) nThreadsPerBlock *= 2;
    int nBlocks = size/4 / nThreadsPerBlock;
    INFO("nThreadsPerBlock = %d, nBlocks = %d\n", nThreadsPerBlock, nBlocks);

    uint32_t* wvalues;
//    wvalues = (uint32_t*)malloc(size);
//    hipHostMalloc(&wvalues, size);
    wvalues = test.addr;
    for(int i = 0; i < size/4; i++) wvalues[i] = i;
//    CUDA_ASSERT(hipHostRegister(wvalues, size, hipHostRegisterDefault));

    // allocate device memory
    uint32_t* values_d;
    INFO("hipMalloc\n");
    hipMalloc(&values_d, size);

    INFO("hipMemcpy\n");
    hipMemcpy(values_d, wvalues, size, hipMemcpyHostToDevice);

    // call kernel
    INFO("kernel1\n");
    kernel1<<<nBlocks, nThreadsPerBlock>>>(values_d);

    // allocate host memory
    uint32_t* rvalues;
    rvalues = (uint32_t*)malloc(size);
//    hipHostMalloc(&rvalues, size);

    // copy values from device to host
    INFO("hipMemcpy\n");
    hipMemcpy(rvalues, values_d, size, hipMemcpyDeviceToHost);

    // check values
    int error = 0;
    for(int i = 0; i < size/4; i++) {
        if(rvalues[i] == ~wvalues[i]) continue;
        error = 1;
        ERR("rvalues[%d] = %d\n", i, rvalues[i]);
    }
    if(error == 0) INFO("OK\n");

    return 0;
}
