#include "hip/hip_runtime.h"

#include "test.h"

inline
void cuda_assert(hipError_t hipError_t, const char* function, const char* file, int line, bool abort = true) {
    if(hipError_t == hipSuccess) return;

    fprintf(stderr, "[%s] %s:%d, hipError_t = %d (%s)\n", function, file, line, hipError_t, hipGetErrorString(hipError_t));
    if(abort) exit(EXIT_FAILURE);
}

#define CUDA_ASSERT(hipError_t) do { cuda_assert((hipError_t), __FUNCTION__, __FILE__, __LINE__); } while(0)

__global__
void kernel1(uint32_t* values) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    values[i] = ~values[i];
}

__host__
int main() {
    int device = 0;
    CUDA_ASSERT(hipSetDevice(device));
    CUDA_ASSERT(hipSetDeviceFlags(hipDeviceMapHost));
    hipDeviceProp_t deviceProperties;
    CUDA_ASSERT(hipGetDeviceProperties(&deviceProperties, device));

    test_t test;
    ssize_t size = test.seek_end(), offset = 0;
    test.mmap(size, offset);

    int nThreadsPerBlock = 1;
    while(2 * nThreadsPerBlock <= deviceProperties.maxThreadsPerBlock) nThreadsPerBlock *= 2;
    int nBlocks = size/4 / nThreadsPerBlock;
    printf("I [] nThreadsPerBlock = %d, nBlocks = %d\n", nThreadsPerBlock, nBlocks);

    uint32_t* wvalues;
//    wvalues = (uint32_t*)malloc(size);
//    hipHostMalloc(&wvalues, size);
    wvalues = test.addr;
    for(int i = 0; i < size/4; i++) wvalues[i] = i;
//    CUDA_ASSERT(hipHostRegister(wvalues, size, hipHostRegisterDefault));

    // allocate device memory
    uint32_t* values_d;
    printf("I [] hipMalloc\n");
    hipMalloc(&values_d, size);

    printf("I [] hipMemcpy\n");
    hipMemcpy(values_d, wvalues, size, hipMemcpyHostToDevice);

    // call kernel
    printf("I [] kernel1\n");
    kernel1<<<nBlocks, nThreadsPerBlock>>>(values_d);

    // allocate host memory
    uint32_t* rvalues;
    rvalues = (uint32_t*)malloc(size);
//    hipHostMalloc(&rvalues, size);

    // copy values from device to host
    hipMemcpy(rvalues, values_d, size, hipMemcpyDeviceToHost);

    // check values
    int error = 0;
    for(int i = 0; i < size/4; i++) {
        if(rvalues[i] == ~wvalues[i]) continue;
        error = 1;
        printf("E [%s] rvalues[%d] = %d\n", __FUNCTION__, i, rvalues[i]);
    }
    if(error == 0) printf("I [%s] OK\n", __FUNCTION__);

    return 0;
}
